#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

extern "C" void matrix_multiply_float(int n, float A[], float B[], float C[])
{
  hipblasStatus_t status;
  hipblasHandle_t handle;
  float *d_A = 0;
  float *d_B = 0;
  float *d_C = 0;
  float alpha = 1.0;
  float beta = 0.0;
  hipMalloc((void **)&d_A, n * n * sizeof(d_A[0]));
  hipMalloc((void **)&d_B, n * n * sizeof(d_B[0]));
  hipMalloc((void **)&d_C, n * n * sizeof(d_C[0]));

  hipblasSetVector(n * n, sizeof(*A), A, 1, d_A, 1);
  hipblasSetVector(n * n, sizeof(*B), B, 1, d_B, 1);
  hipblasSetVector(n * n, sizeof(*C), C, 1, d_C, 1);
  status = hipblasCreate(&handle);
  if (status != HIPBLAS_STATUS_SUCCESS)
  {
    fprintf(stderr, "!!!! CUBLAS initialization error\n");
    return;
  }

  status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, d_A,
                       n, d_B, n, &beta, d_C, n);
  if (status != HIPBLAS_STATUS_SUCCESS)
  {
    fprintf(stderr, "!!!! CUBLAS Sgemm error\n");
    return;
  }

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  status = hipblasDestroy(handle);
  if (status != HIPBLAS_STATUS_SUCCESS)
  {
    fprintf(stderr, "!!!! shutdown error (A)\n");
    return;
  }
}

extern "C" void matrix_multiply_double(int n, double A[], double B[], double C[])
{
  hipblasStatus_t status;
  hipblasHandle_t handle;
  double *d_A = 0;
  double *d_B = 0;
  double *d_C = 0;
  double alpha = 1.0;
  double beta = 0.0;
  hipMalloc((void **)&d_A, n * n * sizeof(d_A[0]));
  hipMalloc((void **)&d_B, n * n * sizeof(d_B[0]));
  hipMalloc((void **)&d_C, n * n * sizeof(d_C[0]));

  hipblasSetVector(n * n, sizeof(*A), A, 1, d_A, 1);
  hipblasSetVector(n * n, sizeof(*B), B, 1, d_B, 1);
  hipblasSetVector(n * n, sizeof(*C), C, 1, d_C, 1);
  status = hipblasCreate(&handle);
  if (status != HIPBLAS_STATUS_SUCCESS)
  {
    fprintf(stderr, "!!!! CUBLAS initialization error\n");
    return;
  }

  status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, d_A,
                       n, d_B, n, &beta, d_C, n);
  if (status != HIPBLAS_STATUS_SUCCESS)
  {
    fprintf(stderr, "!!!! CUBLAS Dgemm error\n");
    return;
  }

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  status = hipblasDestroy(handle);
  if (status != HIPBLAS_STATUS_SUCCESS)
  {
    fprintf(stderr, "!!!! shutdown error (A)\n");
    return;
  }
}
